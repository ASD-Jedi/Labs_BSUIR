#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <ctime>

__global__ void arraySort(int *a)
{
	int idx = threadIdx.x;
	if (a[idx + 1] > a[idx])
	{
		int buffer = a[idx];
		a[idx] = a[idx + 1];
		a[idx + 1] = buffer;
	}
}

int main()
{
	int ha[100000], hv[100000];
	
	int size = sizeof(int) * 100000;
	
	srand(time(0));

	for (int i = 0; i < 100000; i++)
		ha[i] = rand() % 187459356 + 134;

	int *dm;

	hipMalloc((void**) & dm, size);

	hipMemcpy(dm, ha, size, hipMemcpyKind::hipMemcpyHostToDevice);

	arraySort << <20, 100000 >> >(dm);

	hipMemcpy(dm, hv, size, hipMemcpyKind::hipMemcpyDeviceToHost);

	for (int i = 0; i < 100000; i++)
		printf("%d\t", hv[i]);

	getchar();
	return 0;

}