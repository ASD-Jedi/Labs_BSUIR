
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addArrays(int *a, int *b, int *c)
{
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main()
{
	int ha[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10 };
	int hb[] = { 5, 3, 2, 3, 1, 2, 6, 7, 8, 50 };
	int hc[10];

	int *da, *db, *dc;

	int size = sizeof(int)*10;

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);
	hipMalloc((void**)&dc, size);

	hipMemcpy(da, ha, size, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(db, hb, size, hipMemcpyKind::hipMemcpyHostToDevice);

	addArrays <<<1, 10 >>>(da, db, dc);	
	

	hipMemcpy(hc, dc, size, hipMemcpyKind::hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
		printf("%d\n", hc[i]);

	return 0;
}