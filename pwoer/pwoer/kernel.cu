#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__global__ void bubbleMove(int *array_device, int N, int step){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx<(N - 1)) {
		if (step - 2 >= idx){
			if (array_device[idx]<array_device[idx + 1]){
				int buffer = array_device[idx];
				array_device[idx] = array_device[idx + 1];
				array_device[idx + 1] = buffer;
			}
		}
	}
}

void bubleSortCUDA(int *array_host, int N, int blockSize){
	int *array_device; hipMalloc((void **)&array_device, N * sizeof(int));
	for (int i = 0; i < N; i++) array_host[i] = i;
	hipMemcpy(array_device, array_host, N*sizeof(int), hipMemcpyHostToDevice);
	int nblocks = N / blockSize + 1;
	for (int step = 0; step <= N + N; step++) {
		bubbleMove << <nblocks, blockSize >> >(array_device, N, step);
		hipDeviceSynchronize();
	}
	hipMemcpy(array_host, array_device, N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(array_device);
}

void bubleSortCPU(int *array_host, int N){
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N - i - 1; j++) {
			if (array_host[j]<array_host[j + 1]){
				int buffer = array_host[j];
				array_host[j] = array_host[j + 1];
				array_host[j + 1] = buffer;
			}
		}
	}
}

int checkArray(int *array_host, int N){
	int good = 1;
	for (int i = 0; i < N - 1; i++) if (array_host[i]<array_host[i + 1]) { good = 0; printf("i=%d a=%d\n", i, array_host[i]); }
	return good;
}

float measureCUDA(int N, int blockSize){
	int *array_host = (int *)malloc(N * sizeof(int));
	for (int i = 0; i < N; i++) array_host[i] = i;
	clock_t start = clock();
	bubleSortCUDA(array_host, N, blockSize);
	clock_t end = clock();
	if (checkArray(array_host, N) == 1){
		free(array_host);
		return (float)(end - start) / CLOCKS_PER_SEC;
	}
	else {
		free(array_host);
		return -1;
	}
}

float measureCPU(int N) {
	int *array_host = (int *)malloc(N * sizeof(int));
	for (int i = 0; i < N; i++) array_host[i] = i;
	clock_t start = clock();
	bubleSortCPU(array_host, N);
	clock_t end = clock();
	if (checkArray(array_host, N) == 1){
		free(array_host);
		return (float)(end - start) / CLOCKS_PER_SEC;
	}
	else {
		free(array_host);
		return -1;
	}
}

int main(int argc, char const *argv[]) {
	for (int i = 1; i < 10000000; i *= 2) printf("%d %f\t%f\n", i, measureCUDA(i, 256), measureCPU(i));
	return 0;
}
